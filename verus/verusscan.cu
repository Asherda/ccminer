#include "hip/hip_runtime.h"
/**
 * Equihash solver interface for ccminer (compatible with linux and windows)
 * Solver taken from nheqminer, by djeZo (and NiceHash)
 * tpruvot - 2017 (GPL v3)
 */
#include <stdio.h>
#include <unistd.h>
#include <assert.h>

#include <stdexcept>
#include <vector>

#include <sph/sph_sha2.h>

//#include "eqcuda.hpp"
//#include "equihash.h" // equi_verify()

#include <miner.h>
extern "C"
{
#include "./verus/haraka.h"
}

// input here is 140 for the header and 1344 for the solution (equi.cpp)


#include <cuda_helper.h>

#define EQNONCE_OFFSET 30 /* 27:34 */
#define NONCE_OFT EQNONCE_OFFSET

static bool init[MAX_GPUS] = { 0 };
static int valid_sols[MAX_GPUS] = { 0 };
static uint8_t _ALIGN(64) data_sols[MAX_GPUS][10][1536] = { 0 }; // 140+3+1344 required
extern void verus_hash(int thr_id, uint32_t threads, uint32_t startNonce, uint32_t* resNonces);
extern void verus_setBlock(void *blockf,const void *pTargetIn);
extern void verus_init(int thr_id);

#ifndef htobe32
#define htobe32(x) swab32(x)
#endif

extern "C" void VerusHashHalf(uint8_t *result, uint8_t *data, size_t len)
{
    unsigned char buf[128];
    unsigned char *bufPtr = buf;
    int pos = 0, nextOffset = 64;
    unsigned char *bufPtr2 = bufPtr + nextOffset;
    unsigned char *ptr = (unsigned char *)data;
    uint32_t count = 0;

    // put our last result or zero at beginning of buffer each time
    memset(bufPtr, 0, 32);

    // digest up to 32 bytes at a time
    for ( ; pos < len; pos += 32)
    {
        if (len - pos >= 32)
        {
            memcpy(bufPtr + 32, ptr + pos, 32);
        }
        else
        {
            int i = (int)(len - pos);
            memcpy(bufPtr + 32, ptr + pos, i);
            memset(bufPtr + 32 + i, 0, 32 - i);
        }

        count++;

        if (count == 47) break; // exit from cycle before last iteration

        //printf("[%02d.1] ", count); for (int z=0; z<64; z++) printf("%02x", bufPtr[z]); printf("\n");
		haraka512_port_zero(bufPtr2, bufPtr); // ( out, in)
        bufPtr2 = bufPtr;
        bufPtr += nextOffset;
        //printf("[%02d.2] ", count); for (int z=0; z<64; z++) printf("%02x", bufPtr[z]); printf("\n");


        nextOffset *= -1;
    }
    memcpy(result, bufPtr, 32);
};

static void cb_hashdone(int thr_id) {
	if (!valid_sols[thr_id]) valid_sols[thr_id] = -1;
}
static bool cb_cancel(int thr_id) {
	if (work_restart[thr_id].restart)
		valid_sols[thr_id] = -1;
	return work_restart[thr_id].restart;
}

extern "C" int scanhash_verus(int thr_id, struct work *work, uint32_t max_nonce, unsigned long *hashes_done)
{
	uint32_t _ALIGN(64) endiandata[35];
	int i;
	
	
	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;
    int dev_id = device_map[thr_id];
	uint32_t throughput;
	struct timeval tv_start, tv_end, diff;
	double secs, solps;
	
    uint8_t blockhash_half[64];
	uint32_t nonce_buf = 0;
	
    unsigned char block_41970[] = {0xfd, 0x40, 0x05}; // solution
	uint8_t _ALIGN(64) full_data[140+3+1344] = { 0 };
    uint8_t* sol_data = &full_data[140];
	uint32_t intensity = 25;
	
		
	throughput = cuda_default_throughput(thr_id, 1U << intensity);
	if (init[thr_id]) throughput = min(throughput, 0x8000000);
	
	
	memcpy(endiandata, pdata, 140);
	memcpy(full_data, endiandata, 140);  //pdata
    memcpy(full_data +140, block_41970, 3);
  
	
	if (opt_benchmark)
		ptarget[7] = 0x000f;
	if (!init[thr_id])
	{
		hipSetDevice(dev_id);
		if (opt_cudaschedule == -1 && gpu_threads == 1) {
			hipDeviceReset();
			// reduce cpu usage
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
			CUDA_LOG_ERROR();
		}
		cuda_get_arch(thr_id);
	//	api_set_throughput(thr_id, throughput);
		gpulog(LOG_INFO, thr_id, "Intensity set to %g, %u cuda threads", throughput2intensity(throughput), throughput);
		verus_init(thr_id);
		init[thr_id] = true;
	}
	
	VerusHashHalf(blockhash_half, full_data, 1487);	
	memset(blockhash_half + 32, 0x00, 32);
	
	gettimeofday(&tv_start, NULL);  //get millisecond timer val for cal of h
	
	work->valid_nonces = 0;
	verus_setBlock(blockhash_half, work->target); //set data to gpu kernel
	
        
	do {
		
            *hashes_done = (uint64_t)nonce_buf + (uint64_t)throughput;
		     verus_hash(thr_id, throughput, nonce_buf , work->nonces);
			
			if (work->nonces[0] != UINT32_MAX )
		   {
				const uint32_t Htarg = ptarget[7];
				uint32_t _ALIGN(64) vhash[8];
				
                *((uint32_t *)full_data + 368) = work->nonces[0];
                                       
                memset(blockhash_half + 32, 0x0, 32);
                memcpy(blockhash_half + 32, full_data + 1486 - 14, 15);
			//	for (int i = 0; i < 32; i++) printf("", blockhash_half[i]);
				//Sleep(2);
                haraka512_port_zero((unsigned char*)vhash, (unsigned char*)blockhash_half);
				//for (int i = 0; i < 32; i++) printf("", ((uint8_t*)(&vhash))[i]);
				//Sleep(2);
				
				if (vhash[7] <= Htarg && fulltest(vhash, ptarget))
					{
					
					    work->valid_nonces++;
					
                        memcpy(work->data, endiandata, 140);
                        int nonce = work->valid_nonces-1;
                        memcpy(work->extra, sol_data, 1347);
                        bn_store_hash_target_ratio(vhash, work->target, work, nonce);
                                    
						work->nonces[work->valid_nonces - 1] = endiandata[NONCE_OFT];
                        pdata[NONCE_OFT] = endiandata[NONCE_OFT] + 1;
						goto out; 
					}
						
			}
			if ((uint64_t)throughput + (uint64_t)nonce_buf >= (uint64_t)UINT32_MAX) {
				
				break;
			}
		nonce_buf += throughput;

	} while (!work_restart[thr_id].restart);
        
        
out:
	gettimeofday(&tv_end, NULL);
	timeval_subtract(&diff, &tv_end, &tv_start);
	secs = (1.0 * diff.tv_sec) + (0.000001 * diff.tv_usec);
	solps = (double)nonce_buf / secs;
	//gpulog(LOG_INFO, thr_id, "%u K/hashes in %.2f s (%.2f MH/s)", nonce_buf/1000, secs, solps / 1000000);

	return work->valid_nonces;
}

// cleanup
void free_verushash(int thr_id)
{
	if (!init[thr_id])
		return;

	

	init[thr_id] = false;
}


