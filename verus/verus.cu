#include "hip/hip_runtime.h"

#include <miner.h>
extern "C" {
#include <stdint.h>
#include <memory.h>
}
#define HARAKAS_RATE 32
#include <cuda_helper.h>
#define NPT 2
#define NBN 2
__device__  uint32_t sbox[64] =
{ 0x7b777c63, 0xc56f6bf2, 0x2b670130, 0x76abd7fe, 0x7dc982ca, 0xf04759fa, 0xafa2d4ad, 0xc072a49c, 0x2693fdb7, 0xccf73f36, 0xf1e5a534, 0x1531d871, 0xc323c704, 0x9a059618, 0xe2801207, 0x75b227eb, 0x1a2c8309, 0xa05a6e1b, 0xb3d63b52, 0x842fe329, 0xed00d153, 0x5bb1fc20, 0x39becb6a, 0xcf584c4a, 0xfbaaefd0, 0x85334d43, 0x7f02f945, 0xa89f3c50, 0x8f40a351, 0xf5389d92, 0x21dab6bc, 0xd2f3ff10, 0xec130ccd, 0x1744975f, 0x3d7ea7c4, 0x73195d64, 0xdc4f8160, 0x88902a22, 0x14b8ee46, 0xdb0b5ede, 0x0a3a32e0, 0x5c240649, 0x62acd3c2, 0x79e49591, 0x6d37c8e7, 0xa94ed58d, 0xeaf4566c, 0x08ae7a65, 0x2e2578ba, 0xc6b4a61c, 0x1f74dde8, 0x8a8bbd4b, 0x66b53e70, 0x0ef60348, 0xb9573561, 0x9e1dc186, 0x1198f8e1, 0x948ed969, 0xe9871e9b, 0xdf2855ce, 0x0d89a18c, 0x6842e6bf, 0x0f2d9941, 0x16bb54b0 };
#define XT(x) (((x) << 1) ^ ((((x) >> 7) & 1) * 0x1b))
__global__ void verus_gpu_hash(uint32_t threads, uint32_t startNonce, uint32_t *resNonce);
__device__ void haraka512_perm(unsigned char *out, unsigned char *in);
static uint32_t *d_nonces[MAX_GPUS];
__constant__ uint8_t blockhash_half[128];
__constant__ uint32_t ptarget[8];

__device__   void memcpy_decker(unsigned char *dst, unsigned char *src, int len) {
	int i;
	for (i = 0; i< len; i++) { dst[i] = src[i]; }
}

__host__
void verus_init(int thr_id)
{
CUDA_SAFE_CALL(hipMalloc(&d_nonces[thr_id], 2 * sizeof(uint32_t)));
};
void verus_setBlock(void *blockf, const void *pTargetIn)
{
CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(ptarget), pTargetIn, 8 * sizeof(uint32_t), 0, hipMemcpyHostToDevice));
CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(blockhash_half), blockf, 64 * sizeof(uint8_t), 0, hipMemcpyHostToDevice));
};

__host__
void verus_hash(int thr_id, uint32_t threads, uint32_t startNonce, uint32_t *resNonces)
{
	hipMemset(d_nonces[thr_id], 0xff, 2 * sizeof(uint32_t));
	const uint32_t threadsperblock = 256;

	dim3 grid((threads + threadsperblock - 1) / threadsperblock);
	dim3 block(threadsperblock);

	verus_gpu_hash << <grid, block >> >(threads, startNonce, d_nonces[thr_id]);
	//hipDeviceSynchronize();
	hipMemcpy(resNonces, d_nonces[thr_id], NBN * sizeof(uint32_t), hipMemcpyDeviceToHost);
	//memcpy(resNonces, h_nonces[thr_id], NBN * sizeof(uint32_t));

};



//__constant__ static const

// Simulate _mm_aesenc_si128 instructions from AESNI
__device__   void aesenc(unsigned char *s,uint32_t *sharedMemory1)
{
	uint32_t i, t, u;
	register uint32_t  v[4][4];
	
	for (i = 0; i < 16; ++i) {
		v[((i >> 2) + 4 - (i & 3)) & 3][i & 3] = ((uint8_t*)&sharedMemory1[0])[s[i]];
	}

	for (i = 0; i < 4; ++i) {
		t = v[i][0];
		u = v[i][0] ^ v[i][1] ^ v[i][2] ^ v[i][3];
		v[i][0] = v[i][0] ^ u ^ XT(v[i][0] ^ v[i][1]);
		v[i][1] = v[i][1] ^ u ^ XT(v[i][1] ^ v[i][2]);
		v[i][2] = v[i][2] ^ u ^ XT(v[i][2] ^ v[i][3]);
		v[i][3] = v[i][3] ^ u ^ XT(v[i][3] ^ t);
	}
	for (i = 0; i < 16; ++i) {
		s[i] = (unsigned char)v[i >> 2][i & 3]; // VerusHash have 0 rc vector
	}
}

// Simulate _mm_unpacklo_epi32
__device__ __forceinline__   void unpacklo32(unsigned char *t, unsigned char *a, unsigned char *b)
{
	unsigned char tmp[16];
	memcpy_decker(tmp, a, 4);
	memcpy_decker(tmp + 4, b, 4);
	memcpy_decker(tmp + 8, a + 4, 4);
	memcpy_decker(tmp + 12, b + 4, 4);
	memcpy_decker(t, tmp, 16);
}

// Simulate _mm_unpackhi_epi32
__device__  __forceinline__  void unpackhi32(unsigned char *t, unsigned char *a, unsigned char *b)
{
	unsigned char tmp[16];
	memcpy_decker(tmp, a + 8, 4);
	memcpy_decker(tmp + 4, b + 8, 4);
	memcpy_decker(tmp + 8, a + 12, 4);
	memcpy_decker(tmp + 12, b + 12, 4);
	memcpy_decker(t, tmp, 16);

}


__global__ __launch_bounds__(256, 2)
void verus_gpu_hash(uint32_t threads, uint32_t startNonce, uint32_t *resNonce)
{
	uint32_t thread = blockDim.x * blockIdx.x + threadIdx.x;
	if (thread > threads)
		asm("exit;");
	int i, j; unsigned char s[64], tmp[16];
		__shared__ uint32_t sharedMemory1[64];
	if (threadIdx.x < 64)
		sharedMemory1[threadIdx.x] = sbox[threadIdx.x];//	for (i = 0; i < 64; ++i)
					

		uint32_t nounce = startNonce + thread;
		unsigned char in[64];
		
		uint64_t blockhash[4];
		memcpy(s, blockhash_half, 32);
		memset(s + 32, 0x0, 32);
		((uint32_t *)&s)[8] = nounce;
		memcpy(in +48, s + 48, 8);
		//memcpy_decker(s, in, 64);
    #pragma unroll 
		for (i = 0; i < 5; ++i) {
			// aes round(s)
			//__syncthreads();
			for (j = 0; j < 2; ++j) {

				aesenc(s, sharedMemory1);
				aesenc(s + 16, sharedMemory1);
				aesenc(s + 32, sharedMemory1);
				aesenc(s + 48, sharedMemory1);
			}
			unpacklo32(tmp, s, s + 16);
			unpackhi32(s, s, s + 16);
			unpacklo32(s + 16, s + 32, s + 48);
			unpackhi32(s + 32, s + 32, s + 48);
			unpacklo32(s + 48, s, s + 32);
			unpackhi32(s, s, s + 32);
			unpackhi32(s + 32, s + 16, tmp);
			unpacklo32(s + 16, s + 16, tmp);

		}
		for (i = 48; i < 56; i++) {
			s[i] = s[i] ^ in[i];
		}

		memcpy_decker((unsigned char*)blockhash + 24, s + 48, 8);
		
		

		if (blockhash[3] < ((uint64_t*)&ptarget)[3]) { resNonce[0] = nounce; }
	
};


