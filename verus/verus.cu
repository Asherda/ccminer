#include "hip/hip_runtime.h"

#include <miner.h>
extern "C" {
#include <stdint.h>
#include <memory.h>
}
#define HARAKAS_RATE 32

#include <cuda_helper.h>

#define NPT 2
#define NBN 2


__global__ void verus_gpu_hash(uint32_t threads, uint32_t startNonce, uint32_t *resNonce);

__device__ void haraka512_full(unsigned char *out, const unsigned char *in);
__device__ void haraka512_perm(unsigned char *out, const unsigned char *in);
	

static uint32_t *d_nonces[MAX_GPUS];

__constant__ uint8_t blockhash_half[128];
__constant__ uint32_t ptarget[8];

__host__
void verus_init(int thr_id)
{
	
	CUDA_SAFE_CALL(hipMalloc(&d_nonces[thr_id], 2*sizeof(uint32_t)));
   
};


void verus_setBlock(void *blockf,const void *pTargetIn) 
{
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(ptarget), pTargetIn, 8*sizeof(uint32_t), 0, hipMemcpyHostToDevice));
 	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(blockhash_half), blockf, 64*sizeof(uint8_t), 0, hipMemcpyHostToDevice));
	
        
};
__host__ 
void verus_hash(int thr_id, uint32_t threads, uint32_t startNonce, uint32_t *resNonces)
{
	hipMemset(d_nonces[thr_id], 0xff, 2 * sizeof(uint32_t));
	const uint32_t threadsperblock = 256;

	dim3 grid((threads + threadsperblock - 1) / threadsperblock);
	dim3 block(threadsperblock);

	verus_gpu_hash<<<grid, block>>>(threads, startNonce, d_nonces[thr_id]);
	hipDeviceSynchronize();
	hipMemcpy(resNonces, d_nonces[thr_id], 2*sizeof(uint32_t), hipMemcpyDeviceToHost);

	
 
};
__global__ 
void verus_gpu_hash(uint32_t threads, uint32_t startNonce, uint32_t *resNonce)
{
	

	uint32_t thread = blockDim.x * blockIdx.x + threadIdx.x;
	if (thread < threads)
	{uint32_t nounce = startNonce + thread;

    uint8_t hash_buf[64];
    uint8_t blockhash[64];
    
    memcpy(hash_buf,blockhash_half,64);
    memset(hash_buf + 32, 0x0,32);
    //memcpy(hash_buf + 32, (unsigned char *)&full_data + 1486 - 14, 15);
    ((uint32_t *)&hash_buf)[8] = nounce;
  
    
    haraka512_full((unsigned char*)blockhash, (unsigned char*)hash_buf); // ( out, in)

		if (((uint64_t*)&blockhash)[3] < ((uint64_t*)&ptarget)[3]) { resNonce[0] = nounce;}   
    }
};

__device__ void memcpy_decker(unsigned char *dst, unsigned char *src, int len) {
    int i;
    for (i=0; i<len; i++) { dst[i] = src[i]; }
}




//__constant__ static const
__device__  unsigned char sbox[256] =
{ 0x63, 0x7c, 0x77, 0x7b, 0xf2, 0x6b, 0x6f, 0xc5, 0x30, 0x01, 0x67, 0x2b, 0xfe,
  0xd7, 0xab, 0x76, 0xca, 0x82, 0xc9, 0x7d, 0xfa, 0x59, 0x47, 0xf0, 0xad, 0xd4,
  0xa2, 0xaf, 0x9c, 0xa4, 0x72, 0xc0, 0xb7, 0xfd, 0x93, 0x26, 0x36, 0x3f, 0xf7,
  0xcc, 0x34, 0xa5, 0xe5, 0xf1, 0x71, 0xd8, 0x31, 0x15, 0x04, 0xc7, 0x23, 0xc3,
  0x18, 0x96, 0x05, 0x9a, 0x07, 0x12, 0x80, 0xe2, 0xeb, 0x27, 0xb2, 0x75, 0x09,
  0x83, 0x2c, 0x1a, 0x1b, 0x6e, 0x5a, 0xa0, 0x52, 0x3b, 0xd6, 0xb3, 0x29, 0xe3,
  0x2f, 0x84, 0x53, 0xd1, 0x00, 0xed, 0x20, 0xfc, 0xb1, 0x5b, 0x6a, 0xcb, 0xbe,
  0x39, 0x4a, 0x4c, 0x58, 0xcf, 0xd0, 0xef, 0xaa, 0xfb, 0x43, 0x4d, 0x33, 0x85,
  0x45, 0xf9, 0x02, 0x7f, 0x50, 0x3c, 0x9f, 0xa8, 0x51, 0xa3, 0x40, 0x8f, 0x92,
  0x9d, 0x38, 0xf5, 0xbc, 0xb6, 0xda, 0x21, 0x10, 0xff, 0xf3, 0xd2, 0xcd, 0x0c,
  0x13, 0xec, 0x5f, 0x97, 0x44, 0x17, 0xc4, 0xa7, 0x7e, 0x3d, 0x64, 0x5d, 0x19,
  0x73, 0x60, 0x81, 0x4f, 0xdc, 0x22, 0x2a, 0x90, 0x88, 0x46, 0xee, 0xb8, 0x14,
  0xde, 0x5e, 0x0b, 0xdb, 0xe0, 0x32, 0x3a, 0x0a, 0x49, 0x06, 0x24, 0x5c, 0xc2,
  0xd3, 0xac, 0x62, 0x91, 0x95, 0xe4, 0x79, 0xe7, 0xc8, 0x37, 0x6d, 0x8d, 0xd5,
  0x4e, 0xa9, 0x6c, 0x56, 0xf4, 0xea, 0x65, 0x7a, 0xae, 0x08, 0xba, 0x78, 0x25,
  0x2e, 0x1c, 0xa6, 0xb4, 0xc6, 0xe8, 0xdd, 0x74, 0x1f, 0x4b, 0xbd, 0x8b, 0x8a,
  0x70, 0x3e, 0xb5, 0x66, 0x48, 0x03, 0xf6, 0x0e, 0x61, 0x35, 0x57, 0xb9, 0x86,
  0xc1, 0x1d, 0x9e, 0xe1, 0xf8, 0x98, 0x11, 0x69, 0xd9, 0x8e, 0x94, 0x9b, 0x1e,
  0x87, 0xe9, 0xce, 0x55, 0x28, 0xdf, 0x8c, 0xa1, 0x89, 0x0d, 0xbf, 0xe6, 0x42,
  0x68, 0x41, 0x99, 0x2d, 0x0f, 0xb0, 0x54, 0xbb, 0x16 };

#define XT(x) (((x) << 1) ^ ((((x) >> 7) & 1) * 0x1b))

// Simulate _mm_aesenc_si128 instructions from AESNI
__device__  void aesenc(unsigned char *s,const unsigned char sharedMemory1[256])
{
    unsigned char i, t, u, v[4][4];
    for (i = 0; i < 16; ++i) {
        v[((i / 4) + 4 - (i%4) ) % 4][i % 4] = sharedMemory1[s[i]];
    }
    for (i = 0; i < 4; ++i) {
        t = v[i][0];
        u = v[i][0] ^ v[i][1] ^ v[i][2] ^ v[i][3];
        v[i][0] ^= u ^ XT(v[i][0] ^ v[i][1]);
        v[i][1] ^= u ^ XT(v[i][1] ^ v[i][2]);
        v[i][2] ^= u ^ XT(v[i][2] ^ v[i][3]);
        v[i][3] ^= u ^ XT(v[i][3] ^ t);
    }
    for (i = 0; i < 16; ++i) {
        s[i] = v[i / 4][i % 4]; // VerusHash have 0 rc vector
    }
}

// Simulate _mm_unpacklo_epi32
__device__ __forceinline__ void unpacklo32(unsigned char *t, unsigned char *a, unsigned char *b)
{
    unsigned char tmp[16];
    memcpy_decker(tmp, a, 4);
    memcpy_decker(tmp + 4, b, 4);
    memcpy_decker(tmp + 8, a + 4, 4);
    memcpy_decker(tmp + 12, b + 4, 4);
    memcpy_decker(t, tmp, 16);
}

// Simulate _mm_unpackhi_epi32
__device__ __forceinline__ void unpackhi32(unsigned char *t, unsigned char *a, unsigned char *b)
{
    unsigned char tmp[16];
    memcpy_decker(tmp, a + 8, 4);
    memcpy_decker(tmp + 4, b + 8, 4);
    memcpy_decker(tmp + 8, a + 12, 4);
    memcpy_decker(tmp + 12, b + 12, 4);
    memcpy_decker(t, tmp, 16);
}



__device__ void haraka512_perm(unsigned char *out, const unsigned char *in) 
{
    int i, j;
	__align__(4) __shared__ unsigned char sharedMemory1[256];
	if (threadIdx.x < 256)
		sharedMemory1[threadIdx.x] = sbox[threadIdx.x];
    unsigned char s[64], tmp[16];
    memcpy_decker(s, (unsigned char *)in, 64);
#pragma unroll
    for (i = 0; i < 5; ++i) {
        // aes round(s)
		
			for (j = 0; j < 2; ++j) {

				aesenc(s, sharedMemory1);
				aesenc(s + 16, sharedMemory1);
				aesenc(s + 32, sharedMemory1);
				aesenc(s + 48, sharedMemory1);
			}
		
		unpacklo32(tmp, s, s + 16);
		
		unpackhi32(s, s, s + 16);
        unpacklo32(s + 16, s + 32, s + 48);
        unpackhi32(s + 32, s + 32, s + 48);
        unpacklo32(s + 48, s, s + 32);
        unpackhi32(s, s, s + 32);
        unpackhi32(s + 32, s + 16, tmp);
if(i<4)
        unpacklo32(s + 16, s + 16, tmp);
    }

    memcpy_decker(out, s, 64);
}

__device__ void haraka512_full(unsigned char *out, const unsigned char *in)
{
    int i;

    //unsigned char out[64];
    haraka512_perm(out, in);

    for (i = 32; i < 40; i++) {
        out[i-16] = out[i] ^ in[i];
    }

     for (i = 48; i < 56; i++) {
        out[i-24] = out[i] ^ in[i];
    }



    /* Truncated */
    //memcpy_decker(out,      out + 8, 8);
    //memcpy_decker(out + 8,  out + 24, 8);
   // memcpy_decker(out + 16, out + 32, 8);
    //memcpy_decker(out + 24, out + 48, 8);
}
